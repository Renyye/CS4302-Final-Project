#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <iostream>

// ------------------------
// 1. 宏定义
// ------------------------
// Row-major 偏移计算: (row, col, ld=leading_dim)
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// 用于将 float 引用 转换成 float4（需要保证地址是 16 字节对齐）
// 在此示例中保持与原 kernel 风格一致
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

// ------------------------
// 2. 核函数声明 (与您给出的相同)
// ------------------------
__global__ void sgemm_V3(
    float * __restrict__ a, 
    float * __restrict__ b, 
    float * __restrict__ c,
    const int M, 
    const int N, 
    const int K) 
{
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[2][BK][BM];
    __shared__ float s_b[2][BK][BN];

    float r_load_a[4];
    float r_load_b[4];
    float r_comp_a[TM];
    float r_comp_b[TN];
    float r_c[TM][TN] = {0.0};

    // 预先计算一些索引
    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    // --- 第一次加载 ---
    {
        int load_a_gmem_k = load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);

        // 从全局内存一次性加载 float4
        FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        // 写到共享内存
        s_a[0][load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[0][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[0][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[0][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];

        FLOAT4(s_b[0][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);
    }

    // --- 主循环 ---
    for (int bk = 1; bk < (K + BK - 1) / BK; bk++) {

        int smem_sel = (bk - 1) & 1;
        int smem_sel_next = bk & 1;

        // 读取下一块 A, B
        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);

        FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        // 对上一次加载的共享内存做计算
        #pragma unroll
        for (int tk = 0; tk < BK; tk++) {
            FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2         ]);
            FLOAT4(r_comp_a[4]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2 + BM / 2]);
            FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2         ]);
            FLOAT4(r_comp_b[4]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2 + BN / 2]);

            #pragma unroll
            for (int tm = 0; tm < TM; tm++) {
                #pragma unroll
                for (int tn = 0; tn < TN; tn++) {
                    r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                }
            }
        }

        // 把下一块加载到共享内存
        s_a[smem_sel_next][load_a_smem_k    ][load_a_smem_m] = r_load_a[0];
        s_a[smem_sel_next][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[smem_sel_next][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[smem_sel_next][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[smem_sel_next][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);

        __syncthreads();
    }

    // --- 处理最后一块 ---
    #pragma unroll
    for (int tk = 0; tk < BK; tk++) {
        FLOAT4(r_comp_a[0]) = FLOAT4(s_a[1][tk][ty * TM / 2         ]);
        FLOAT4(r_comp_a[4]) = FLOAT4(s_a[1][tk][ty * TM / 2 + BM / 2]);
        FLOAT4(r_comp_b[0]) = FLOAT4(s_b[1][tk][tx * TN / 2         ]);
        FLOAT4(r_comp_b[4]) = FLOAT4(s_b[1][tk][tx * TN / 2 + BN / 2]);

        #pragma unroll
        for (int tm = 0; tm < TM; tm++) {
            #pragma unroll
            for (int tn = 0; tn < TN; tn++) {
                r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
            }
        }
    }

    // --- 写回到全局 C ---
    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);

        FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
        FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
    }

    #pragma unroll
    for (int i = 0; i < TM / 2; i++) {
        int store_c_gmem_m = by * BM + BM / 2 + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);

        FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
        FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
    }
}


// ------------------------
// 3. 封装函数
// ------------------------
at::Tensor custom_matrix_mul(
    at::Tensor A,  // [M, K]
    at::Tensor B   // [K, N]
) {
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dim() == 2 && B.dim() == 2, 
                "A, B must be 2D tensors");
    TORCH_CHECK(A.dtype() == torch::kFloat32 && B.dtype() == torch::kFloat32,
                "A, B must be float32");
    TORCH_CHECK(A.size(1) == B.size(0),
                "A.size(1) must match B.size(0), shapes: [M, K], [K, N]");

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    // 创建输出 [M, N]
    auto C = torch::zeros({M, N}, A.options());

    // 获取原始指针
    float* a_ptr = (float*)A.data_ptr<float>();
    float* b_ptr = (float*)B.data_ptr<float>();
    float* c_ptr = (float*)C.data_ptr<float>();

    // 配置 block / grid
    // 与内核的 BM=128, BN=128, TM=8, TN=8 对应
    // blockDim = (BN/TN=16, BM/TM=16)
    dim3 blockDim(16, 16);

    // gridDim = ( (N+BN-1)/BN, (M+BM-1)/BM )
    dim3 gridDim((N + 128 - 1) / 128,
                 (M + 128 - 1) / 128);

    // 调用 kernel
    sgemm_V3<<<gridDim, blockDim>>>(
        a_ptr, b_ptr, c_ptr,
        M, N, K
    );

    // 错误检查
    hipDeviceSynchronize();
    auto err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(err));

    return C;
}
