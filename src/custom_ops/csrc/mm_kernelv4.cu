#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <float.h>
#include <torch/extension.h>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

__global__ void custom_matMul_kernel_v4(
    const float* __restrict__ A,
    const float* __restrict__ B,
    float*       __restrict__ C,
    int M, int N, int K)
{
    // 规定块大小（覆盖C中的子块大小）
    const int BM = 128;   // 每个 block 负责 C 的 128 行
    const int BN = 128;   // 每个 block 负责 C 的 128 列
    const int BK = 8;     // 分块（tile）宽度（K 方向一次处理 8）
    
    // 规定线程在 block 内负责 8x8 大小的输出子块
    const int TM = 8;     // 每个线程在行方向上处理 8 个输出元素
    const int TN = 8;     // 每个线程在列方向上处理 8 个输出元素

    // 计算该 block 在网格 (grid) 中的坐标
    int bx = blockIdx.x;  // block 在 x 方向的索引, 对应输出 C 的"行块"
    int by = blockIdx.y;  // block 在 y 方向的索引, 对应输出 C 的"列块"

    // 计算该线程在 block 内的坐标
    int tx = threadIdx.x; // 线程 x 方向索引
    int ty = threadIdx.y; // 线程 y 方向索引

    // 该 block 在 C 中左上角的全局行/列索引
    int block_start_row = bx * BM;
    int block_start_col = by * BN;

    // 计算要分多少次 tile 才能覆盖完 K
    int num_tiles = (K + BK - 1) / BK;

    // -----------------------------
    // 为 A、B 分配共享内存 (更合理的排布)
    //   s_a:  128 x 8
    //   s_b:    8 x 128
    // -----------------------------
    __shared__ float s_a[128][BK];  // = [128][8]
    __shared__ float s_b[BK][128];  // = [8][128]

    // -----------------------------
    // 每个线程维护一个 8x8 的累加寄存器块
    // -----------------------------
    float r_c[TM][TN];
    #pragma unroll
    for (int i = 0; i < TM; i++)
    {
        #pragma unroll
        for (int j = 0; j < TN; j++)
        {
            r_c[i][j] = 0.0f;
        }
    }

    // -----------------------------
    // 遍历所有的 tile (在K方向)
    // -----------------------------
    for (int tile = 0; tile < num_tiles; tile++)
    {
        // 1) 把 A 的一块 [128 x BK] 加载到 s_a
        //    这里让 (threadIdx.x, threadIdx.y) 通过小循环分担所有元素
        {
            // A 的全局起始: (block_start_row, tile*BK)
            // A 的大小: M x K
            // s_a 的大小: 128 x 8
            // 我们让每个线程通过 2D 循环把需要的元素搬到共享内存

            for (int i = ty; i < 128; i += blockDim.y)    // blockDim.y=16
            {
                for (int j = tx; j < BK;   j += blockDim.x) // blockDim.x=16, BK=8
                {
                    int global_row = block_start_row + i;      // 行
                    int global_col = tile * BK + j;            // 列
                    if (global_row < M && global_col < K)
                        s_a[i][j] = A[ global_row * K + global_col ];
                    else
                        s_a[i][j] = 0.0f;
                }
            }
        }

        // 2) 把 B 的一块 [BK x 128] 加载到 s_b
        {
            // B 的全局起始: (tile*BK, block_start_col)
            // B 的大小: K x N
            // s_b 的大小: 8 x 128
            // 同理，每个线程通过 2D 循环把 B 分块加载进来

            for (int i = ty; i < BK;   i += blockDim.y)    // BK=8
            {
                for (int j = tx; j < 128; j += blockDim.x) // 128
                {
                    int global_row = tile * BK + i;         // 行
                    int global_col = block_start_col + j;    // 列
                    if (global_row < K && global_col < N)
                        s_b[i][j] = B[ global_row * N + global_col ];
                    else
                        s_b[i][j] = 0.0f;
                }
            }
        }

        // 同步，确保共享内存已填充完毕
        __syncthreads();

        // 3) 做本 tile 的乘加运算
        //    对 s_a [128 x 8] 和 s_b [8 x 128]
        //    每个线程要完成 8x8 的结果累加
        {
            // 先确定线程在 s_a、s_b 中读取的行/列范围
            // 每个线程在 C 中对 (ty*TM + i, tx*TN + j) 那些行列负责
            // 逐列 (k) 做乘加
            #pragma unroll
            for (int k = 0; k < BK; k++)
            {
                #pragma unroll
                for (int i = 0; i < TM; i++)
                {
                    float a_val = s_a[ ty*TM + i ][k];
                    #pragma unroll
                    for (int j = 0; j < TN; j++)
                    {
                        r_c[i][j] += a_val * s_b[k][ tx*TN + j ];
                    }
                }
            }
        }

        // 同步，为下一次 tile 加载做准备
        __syncthreads();
    }

    // -----------------------------
    // 写回结果到全局内存
    // -----------------------------
    for (int i = 0; i < TM; i++)
    {
        int c_row = block_start_row + ty*TM + i;
        for (int j = 0; j < TN; j++)
        {
            int c_col = block_start_col + tx*TN + j;
            if (c_row < M && c_col < N)
            {
                C[c_row * N + c_col] = r_c[i][j];
            }
        }
    }
}


at::Tensor custom_matrix_mul_v4(at::Tensor A, at::Tensor B) {
    // 检查输入张量在CUDA上且是float类型
    TORCH_CHECK(A.is_cuda(), "Input tensor A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "Input tensor B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "Input tensor A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "Input tensor B must be float32");
    TORCH_CHECK(A.size(1) == B.size(0), "Inner dimensions must match for matMul");

    // 获取张量维度
    const int M = A.size(0);
    const int K = A.size(1);
    const int N = B.size(1);

    // 创建输出张量
    auto C = torch::zeros({M, N}, torch::device(torch::kCUDA).dtype(torch::kFloat32));

    // 获取设备数据指针
    float* d_A = A.data_ptr<float>();
    float* d_B = B.data_ptr<float>();
    float* d_C = C.data_ptr<float>();

    // 设置 CUDA 核函数的线程块和网格大小
    const int BM = 128; // 每个 block 处理的行数
    const int BN = 128; // 每个 block 处理的列数
    dim3 blockDim(16, 16); // 每个线程块的线程数
    dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM); // 网格大小

    // 调用 CUDA 核函数
    custom_matMul_kernel_v4<<<gridDim, blockDim>>>(
        d_A, d_B, d_C, M, N, K
    );

    // 检查 CUDA 内核是否正确执行
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    // 返回结果张量
    return C;
}