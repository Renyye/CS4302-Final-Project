#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <torch/extension.h>

// 定义分块大小和线程大小
#define TILE_SIZE_M 16
#define TILE_SIZE_P 16
#define TILE_SIZE_K 16


__global__ void bmm_kernel(const float *d_A, const float *d_B, float *d_C,
                                      int batch_size, int M, int N, int P) {
    // 计算当前批次、行和列
    int batch = blockIdx.z;
    int row = blockIdx.y * TILE_SIZE_M + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE_P + threadIdx.x;

    // 初始化累加器
    float sum = 0.0f;

    // 分块遍历K维度
    for (int tile = 0; tile < (N + TILE_SIZE_K - 1) / TILE_SIZE_K; ++tile) {
        // 声明共享内存
        __shared__ float As[TILE_SIZE_M][TILE_SIZE_K];
        __shared__ float Bs[TILE_SIZE_K][TILE_SIZE_P];

        // 计算A和B的全局索引
        int A_row = row;
        int A_col = tile * TILE_SIZE_K + threadIdx.x;
        int B_row = tile * TILE_SIZE_K + threadIdx.y;
        int B_col = col;

        // 加载A和B到共享内存
        if (A_row < M && A_col < N) {
            As[threadIdx.y][threadIdx.x] = d_A[batch * M * N + A_row * N + A_col];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (B_row < N && B_col < P) {
            Bs[threadIdx.y][threadIdx.x] = d_B[batch * N * P + B_row * P + B_col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // 同步线程，确保共享内存中的数据已经加载完成
        __syncthreads();

        // 计算部分和
        for (int k = 0; k < TILE_SIZE_K; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        // 同步线程，确保所有线程都完成了部分和的计算
        __syncthreads();
    }

    // 写回结果
    if (row < M && col < P && batch < batch_size) {
        d_C[batch * M * P + row * P + col] = sum;
    }
}

__global__ void shared_weight_bmm_kernel(const float *d_A, const float *d_B, float *d_C,
                                      int batch_size, int M, int N, int P) {
    // 计算当前批次、行和列
    int batch = blockIdx.z;
    int row = blockIdx.y * TILE_SIZE_M + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE_P + threadIdx.x;

    // 初始化累加器
    float sum = 0.0f;

    // 分块遍历K维度
    for (int tile = 0; tile < (N + TILE_SIZE_K - 1) / TILE_SIZE_K; ++tile) {
        __shared__ float As[TILE_SIZE_M][TILE_SIZE_K];
        __shared__ float Bs[TILE_SIZE_K][TILE_SIZE_P];

        // 加载A和B到共享内存
        int A_row = row;
        int A_col = tile * TILE_SIZE_K + threadIdx.x;
        int B_row = tile * TILE_SIZE_K + threadIdx.y;
        int B_col = col;

        As[threadIdx.y][threadIdx.x] = (A_row < M && A_col < N) ? d_A[batch * M * N + A_row * N + A_col] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] = (B_row < N && B_col < P) ? d_B[B_row * P + B_col] : 0.0f;

        __syncthreads();

        // 计算部分和
        for (int k = 0; k < TILE_SIZE_K; ++k) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // 写回结果
    if (row < M && col < P && batch < batch_size) {
        d_C[batch * M * P + row * P + col] = sum;
    }
}


// 封装函数：接收PyTorch张量并调用kernel
at::Tensor custom_bmm(at::Tensor A, at::Tensor B) {
    // 假设输入为 (batch_size, M, N), (batch_size, N, P)
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "B must be float32");

    int batch_size = A.size(0);
    int M = A.size(1);
    int N = A.size(2);
    int P = B.size(2);

    // 创建输出tensor (batch_size, M, P)
    auto C = torch::zeros({batch_size, M, P}, torch::dtype(torch::kFloat32).device(A.device()));

    // 获取raw pointers
    float *d_A = A.data_ptr<float>();
    float *d_B = B.data_ptr<float>();
    float *d_C = C.data_ptr<float>();

    dim3 blockDim(16, 16, 1);
    dim3 gridDim((P + blockDim.x - 1)/blockDim.x,
                (M + blockDim.y - 1)/blockDim.y,
                batch_size);

    // 调用kernel
    bmm_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, batch_size, M, N, P);

    // 同步和错误检查
    hipDeviceSynchronize();
    auto err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return C;
}

at::Tensor shared_weight_bmm(at::Tensor A, at::Tensor B) {
    // 检查输入张量
    TORCH_CHECK(A.is_cuda(), "A 必须是 CUDA 张量");
    TORCH_CHECK(B.is_cuda(), "B 必须是 CUDA 张量");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "A 必须是 float32 类型");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "B 必须是 float32 类型");
    TORCH_CHECK(A.dim() == 3, "A 必须是 3 维张量 [B, M, N]");
    TORCH_CHECK(B.dim() == 2, "B 必须是 2 维张量 [N, P]");
    // 确保权重 B 的形状与 A 的最后一个维度匹配
    TORCH_CHECK(A.size(2) == B.size(0), "A 的 N 维度必须等于 B 的 N 维度");

    int batch_size = A.size(0);
    int M = A.size(1);
    int N = A.size(2);
    int P = B.size(1);

    // 创建输出张量 [B, M, P]
    auto C = torch::zeros({batch_size, M, P}, A.options());

    // 获取原始指针
    const float *d_A_ptr = A.data_ptr<float>();
    const float *d_B_ptr = B.data_ptr<float>();
    float *d_C_ptr = C.data_ptr<float>();

    // 定义 block 和 grid 维度
    dim3 blockDim(TILE_SIZE_P, TILE_SIZE_M, 1);
    dim3 gridDim(
        (P + TILE_SIZE_P - 1) / TILE_SIZE_P,
        (M + TILE_SIZE_M - 1) / TILE_SIZE_M,
        batch_size
    );

    // 调用 CUDA 内核
    shared_weight_bmm_kernel<<<gridDim, blockDim>>>(d_A_ptr, d_B_ptr, d_C_ptr, batch_size, M, N, P);

    // 检查 CUDA 错误
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        // 抛出异常以通知 PyTorch
        TORCH_CHECK(false, "CUDA kernel failed");
    }

    return C;
}
