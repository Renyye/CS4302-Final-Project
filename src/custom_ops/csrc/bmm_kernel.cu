#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <torch/extension.h>

__global__ void bmm_kernel(float *d_A, float *d_B, float *d_C, int batch_size, int M, int N, int P) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int batch = blockIdx.z * blockDim.z + threadIdx.z;
    // printf d_B
    if (batch < batch_size && row < M && col < P) {
        printf("Batch: %d, Row: %d, Col: %d, A[Batch=%d][Row=%d][i=%d]=%f, B[Batch=%d][i=%d][Col=%d]=%f\n",
                batch, row, col,
                batch, row, 0, d_A[batch * M * N + row * N + 0],
                batch, 0, col, d_B[batch * N * P + 0 * P + col]);
    }
    if (batch < batch_size && row < M && col < P) {
        float sum = 0.0f;
        
                for (int i = 0; i < N; ++i) {
                        sum += d_A[batch * M * N + row * N + i] * d_B[batch * N * P + i * P + col];
                        printf("Batch: %d, Row: %d, Col: %d, i: %d, A[Batch=%d][Row=%d][i=%d]=%f, B[Batch=%d][i=%d][Col=%d]=%f, Partial Sum: %f\n",
                                batch, row, col, i,
                                batch, row, i, d_A[batch * M * N + row * N + i],
                                batch, i, col, d_B[batch * N * P + i * P + col],
                                sum);
                }
        d_C[batch * M * P + row * P + col] = sum;
                printf("Batch: %d, Row: %d, Col: %d, Sum: %f\n", batch, row, col, sum);
    }
}

// 封装函数：接收PyTorch张量并调用kernel
at::Tensor custom_bmm(at::Tensor A, at::Tensor B) {
    // 假设输入为 (batch_size, M, N), (batch_size, N, P)
    TORCH_CHECK(A.is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "B must be float32");

    int batch_size = A.size(0);
    int M = A.size(1);
    int N = A.size(2);
    int P = B.size(2);

    // 创建输出tensor (batch_size, M, P)
    auto C = torch::zeros({batch_size, M, P}, torch::dtype(torch::kFloat32).device(A.device()));

    // 获取raw pointers
    float *d_A = A.data_ptr<float>();
    float *d_B = B.data_ptr<float>();
    float *d_C = C.data_ptr<float>();

    dim3 blockDim(16, 16, 1);
    dim3 gridDim((P + blockDim.x - 1)/blockDim.x,
                (M + blockDim.y - 1)/blockDim.y,
                batch_size);


    // 调用kernel
    bmm_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, batch_size, M, N, P);

    // 同步和错误检查
    hipDeviceSynchronize();
    auto err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA Error: ", hipGetErrorString(err));

    return C;
}
