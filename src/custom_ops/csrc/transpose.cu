
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void transpose(float *d_A, float *d_T, int M, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// swap elements via transpose
	if (row < M && col < N) {
		d_T[col * M + row] = d_A[row * N + col];
	}
}
